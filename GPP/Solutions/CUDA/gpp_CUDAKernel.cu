#include "hip/hip_runtime.h"
#include "../../../ComplexClass/CustomComplex.h"

__global__ void gpp_2D_CUDAKernel(int number_bands, int ngpown, int ncouls, \
    device_Array1D<dataType> achtemp_re, device_Array1D<dataType> achtemp_im, \
    device_Array1D<int> inv_igp_index, device_Array1D<int> indinv, device_Array1D<dataType> wx_array, \
        device_Array2D<CustomComplex<dataType>> aqsmtemp, device_Array2D<CustomComplex<dataType>> aqsntemp, \
        device_Array1D<dataType> vcoul, device_Array2D<CustomComplex<dataType>> wtilde_array, \
        device_Array2D<CustomComplex<dataType>> I_eps_array)
{
    dataType achtemp_re_loc[nend-nstart], achtemp_im_loc[nend-nstart];
    for(int iw = nstart; iw < nend; ++iw) {achtemp_re_loc[iw] = 0.00; achtemp_im_loc[iw] = 0.00;}

    for(int n1 = blockIdx.x; n1<number_bands; n1+=gridDim.x)
    {
        for(int my_igp=blockIdx.y; my_igp<ngpown; my_igp+=gridDim.y)
        {
            const int indigp = inv_igp_index(my_igp);
            const int igp = indinv(indigp);
            CustomComplex<dataType> sch_store1 = CustomComplex_conj(aqsmtemp(n1,igp))*  aqsntemp(n1,igp) * 0.5 * vcoul(igp);

            for(int ig = threadIdx.x; ig<ncouls; ig+=blockDim.x)
            {
                for(int iw = nstart; iw < nend; ++iw)
                {
                    CustomComplex<dataType> wdiff = wx_array(iw) - wtilde_array(my_igp,ig);
                    double wdiff_r = CustomComplex_real(wdiff * CustomComplex_conj(wdiff));
                    wdiff_r = 1/wdiff_r;

                    CustomComplex<dataType> delw = wtilde_array(my_igp, ig) * CustomComplex_conj(wdiff) * wdiff_r;
                    CustomComplex<dataType> sch_array = sch_store1 * I_eps_array(my_igp,ig) * delw ;

                    achtemp_re_loc[iw] += CustomComplex_real(sch_array);
                    achtemp_im_loc[iw] += CustomComplex_imag(sch_array);
                }
            }
        } //ngpown
    } //number_bands

    //Add the final results here;
    for(int iw = nstart; iw < nend; ++iw)
    {
        atomicAdd(&achtemp_re(iw), achtemp_re_loc[iw]);
        atomicAdd(&achtemp_im(iw), achtemp_im_loc[iw]);
    }
}

void noflagOCC_cudaKernel(int number_bands, int ngpown, int ncouls, \
        device_Array1D<dataType> achtemp_re, device_Array1D<dataType> achtemp_im, \
        device_Array1D<int> inv_igp_index, device_Array1D<int> indinv, device_Array1D<dataType> wx_array, \
        device_Array2D<CustomComplex<dataType>> aqsmtemp, device_Array2D<CustomComplex<dataType>> aqsntemp, \
        device_Array1D<dataType> vcoul, device_Array2D<CustomComplex<dataType>> wtilde_array, \
        device_Array2D<CustomComplex<dataType>> I_eps_array)

{
    dim3 numBlocks(number_bands, ngpown);
    dim3 numThreads(32,1,1);
    printf("Launching a double dimension grid with numBlocks = (%d, %d) and %d threadsPerBlock \n", number_bands, ngpown, numThreads.x);

    gpp_2D_CUDAKernel<<<numBlocks, numThreads>>>(number_bands, ngpown, ncouls, achtemp_re, achtemp_im, \
            inv_igp_index, indinv, wx_array, \
        aqsmtemp, aqsntemp, vcoul, wtilde_array, I_eps_array);
}


